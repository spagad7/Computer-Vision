#include "hip/hip_runtime.h"
#ifndef KERNELS
#define KERNELS

#include "../include/matrix.hpp"
#include <climits>


// Naive implementation of matrix multiplication kernel
__global__ void matmult_kernel_v1(Matrix M, Matrix N, Matrix P)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0;

    if(row < P.rows && col < P.cols)
    {
        for(int i=0; i<M.cols; i++)
            sum += M.elements[row*M.cols + i] * N.elements[i*N.cols + col];

        P.elements[row*P.cols + col] = sum;
    }
}



// Naive implementation of finding minimum element in a row
__global__ void find_min(Matrix M, Match* matches)
{
    int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    float min = INT_MAX;
    int minIdx;

    if(tIdx < M.rows)
    {
        for(int i=0; i<M.cols; i++)
        {
            if(M.elements[tIdx*M.cols + i] < min)
            {
                min = M.elements[tIdx*M.cols + i];
                minIdx = i;
            }
        }

        matches[tIdx].distance = min;
        matches[tIdx].idx1 = tIdx;
        matches[tIdx].idx2 = minIdx;
    }
}


__global__ void dummy()
{
    int tIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int tmp = 0;
}

#endif
