#include "hip/hip_runtime.h"
#include "../include/gpu_matcher.hpp"
#include "kernels.cu"


// Constructor for GPU Matcher
GPUMatcher::GPUMatcher(){}


// Function to perform feature matching on GPU
void GPUMatcher::match(cv::Mat& desc1, cv::Mat& desc2,
                        std::vector<cv::DMatch>& matches)
{
    Matrix desc1_h, desc2_h, product_mat_h;
    Matrix desc1_d, desc2_d, product_mat_d;

    // Transpose desc1 matrix
    cv::Mat desc2_trans;
    cv::transpose(desc2, desc2_trans);

    // Allocate matrices on GPU
    AllocateDeviceMatrix(desc1_d, desc1.rows, desc1.cols);
    AllocateDeviceMatrix(desc2_d, desc2_trans.rows, desc2_trans.cols);
    AllocateDeviceMatrix(product_mat_d, desc1.rows, desc2_trans.cols);

    // Flatten matrices
    FlattenMatrix(desc1, desc1_h);
    FlattenMatrix(desc2_trans, desc2_h);

    // Allocate Matrix on Host
    //AllocateHostMatrix(product_mat_h, desc1.rows, desc2_trans.cols);

    // Copy flattened matrices to device memory
    CopyMatrixToDevice(desc1_h, desc1_d);
    CopyMatrixToDevice(desc2_h, desc2_d);

/*

// TODO: Implement dynamic block size based on product_mat_d size

    // Configure matrix multiplication kernel
    hipDeviceProp_t prop;
    int deviceId = 0;
    hipError_t ret_val = hipGetDeviceProperties(&prop, deviceId);
    if(ret_val != hipSuccess)
    {
        std::cout << "Error getting cuda device property!" << std::endl;
        exit(-1);
    }

    dim3 dimBlock, dimGrid;
    if(product_mat_d.rows*product_mat_d.cols < prop.maxThreadsPerBlock)
    {

    }
*/

    dim3 dimBlock, dimGrid;
    dimBlock.x = 16, dimBlock.y = 16, dimBlock.z=1;
    dimGrid.x = (int)ceil((float)product_mat_d.cols/dimBlock.x);
    dimGrid.y = (int)ceil((float)product_mat_d.rows/dimBlock.y);
    dimGrid.z = 1;

    // Launch matrix multiplication kernel
    matmult_kernel_v1<<<dimGrid, dimBlock>>>(desc1_d, desc2_d, product_mat_d);

    hipDeviceSynchronize();
    // Copy matrix to host for testing
    //CopyMatrixToHost(product_mat_d, product_mat_h);

    // Allocate memory for array of match objects in host and device
    Match *matches_h, *matches_d;
    //AllocateHostMatchArray(matches_h, product_mat_d.rows);
    //AllocateDeviceMatchArray(matches_d, product_mat_d.rows);

    matches_h = (Match *)malloc(product_mat_d.rows * sizeof(Match));
    if(matches_h == NULL)
    {
        std::cout << "Error allocating host memory!" << std::endl;
        exit(-1);
    }

    hipError_t ret_val = hipMalloc((void**)&matches_d, product_mat_d.rows * sizeof(Match));
    if(ret_val != hipSuccess)
    {
        std::cout << "Error allocating memory on device!" << std::endl;
        exit(-1);
    }

    // Configure find_min kernel
    dimBlock.x = 256;
    dimBlock.y = 1;
    dimBlock.z = 1;
    dimGrid.x = (int)ceil((float)product_mat_d.rows/dimBlock.x);
    dimGrid.y = 1;
    dimGrid.z = 1;

    // Launch find_min kernel
    find_min<<<dimGrid, dimBlock>>>(product_mat_d, matches_d);
    //dummy<<<dimGrid, dimBlock>>>();
    hipDeviceSynchronize();

    // Copy Match array from device
    CopyMatchArrayToHost(matches_d, matches_h, product_mat_d.rows);

    // Convert Match array to vector of DMatch objects
    ConvertMatchToDMatch(matches_h, matches, product_mat_d.rows);
}


// Function to allocate matrix on device
void GPUMatcher::AllocateDeviceMatrix(Matrix& M, int rows, int cols)
{
    M.rows = rows;
    M.cols = cols;
    int size = rows * cols * sizeof(float);
    hipError_t ret_val = hipMalloc((void**)&M.elements, size);
    if(ret_val != hipSuccess)
    {
        std::cout << "Error allocating memory on device!" << std::endl;
        exit(-1);
    }
}


// Function to free Matrix in device memory
void GPUMatcher::FreeDeviceMatrix(Matrix& M)
{
    hipError_t ret_val = hipFree(M.elements);
    if(ret_val != hipSuccess)
    {
        std::cout << "Unable to free allocated device memory!" << std::endl;
    }
    M.elements = NULL;
}


// Function to allocate matrix in host memory
void GPUMatcher::AllocateHostMatrix(Matrix& M, int rows, int cols)
{
    M.rows = rows;
    M.cols = cols;
    int size = rows * cols * sizeof(float);

    M.elements = (float *)malloc(size);
    if(M.elements == NULL)
    {
        std::cout << "Error allocating host memory!" << std::endl;
        exit(-1);
    }

}


// Function to allocate array of Match objects in host memory
void GPUMatcher::AllocateHostMatchArray(Match* matches, int size)
{
    matches = (Match *)malloc(size * sizeof(Match));
    if(matches == NULL)
    {
        std::cout << "Error allocating host memory!" << std::endl;
        exit(-1);
    }
}


// Function to allocate array of Match objects in device memory
void GPUMatcher::AllocateDeviceMatchArray(Match* matches, int size)
{
    hipError_t ret_val = hipMalloc((void**)&matches, size * sizeof(Match));
    if(ret_val != hipSuccess)
    {
        std::cout << "Error allocating memory on device!" << std::endl;
        exit(-1);
    }
}


// Function to free device Match array
void GPUMatcher::FreeDeviceMatchArray(Match* matches)
{
    hipError_t ret_val = hipFree(matches);
    if(ret_val != hipSuccess)
    {
        std::cout << "Unable to free allocated device memory!" << std::endl;
    }
    matches = NULL;
}


// Function to flatten 2D matrix
void GPUMatcher::FlattenMatrix(cv::Mat& M_Mat, Matrix& M)
{
    M.rows = M_Mat.rows;
    M.cols = M_Mat.cols;
    M.elements = (float *)malloc(M_Mat.rows * M_Mat.cols * sizeof(float));

    for(int i=0; i<M_Mat.rows; i++)
    {
        for(int j=0; j<M_Mat.cols; j++)
        {
            M.elements[i * M_Mat.cols + j] = M_Mat.at<float>(i,j);
        }
    }
}


// Function to copy matrix from host to device
void GPUMatcher::CopyMatrixToDevice(Matrix& M_h, Matrix& M_d)
{
    int size = M_h.rows * M_h.cols * sizeof(float);
    hipError_t ret_val = hipMemcpy(M_d.elements, M_h.elements, size,
                                    hipMemcpyHostToDevice);
    if(ret_val != hipSuccess)
    {
        std::cout << "Unable to copy data to device memory" << std::endl;
        exit(-1);
    }
}


// Function to copy matrix from device to host
void GPUMatcher::CopyMatrixToHost(Matrix M_d, Matrix M_h)
{
    M_h.rows = M_d.rows;
    M_h.cols = M_d.cols;
    int size = M_d.rows * M_d.cols * sizeof(float);
    hipError_t ret_val = hipMemcpy(M_h.elements, M_d.elements, size,
                                    hipMemcpyDeviceToHost);
    if(ret_val != hipSuccess)
    {
        std::cout << "Unable to copy data fom device to host" << std::endl;
        exit(-1);
    }
}


// Function to copy array of Match objects from device to host memory
void GPUMatcher::CopyMatchArrayToHost(Match* matches_d, Match* matches_h,
                                        int size)
{
    hipError_t ret_val = hipMemcpy(matches_h, matches_d, size*sizeof(Match),
                                        hipMemcpyDeviceToHost);
    if(ret_val != hipSuccess)
    {
        std::cout << "Error copying data from device to host!" << std::endl;
        exit(-1);
    }
}


// Function to copy array of Match objects from device to host memory
void GPUMatcher::CopyMatchArrayToDevice(Match* matches_h, Match* matches_d,
                                        int size)
{
    hipError_t ret_val = hipMemcpy(matches_d, matches_h, size*sizeof(Match),
                                        hipMemcpyHostToDevice);
    if(ret_val != hipSuccess)
    {
        std::cout << "Error copying data from host to device!" << std::endl;
        exit(-1);
    }
}


// Function to conver array of Match objects to vector of cv::DMatch objects
void GPUMatcher::ConvertMatchToDMatch(Match* matches_h,
                                        std::vector<cv::DMatch>& matches,
                                        int size)
{
    for(int i=0; i<size; i++)
    {
        matches.push_back(cv::DMatch(matches_h[i].idx1, matches_h[i].idx2,
                                    0, matches_h[i].distance));
    }
}
